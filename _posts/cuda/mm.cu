#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 10000, M 2000

__global__ void sgemm(const float* a, const float* b, float* c, int row, int col, int k) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < row && j < col) {
        float sum = 0.0f;
        for (int index = 0; index < k; index++) {
            sum += a[i * row + k] * B[k * index + j];
        }
        c[i * row + col] = sum;
    }
}

int main() {
    float *hA, *hB, *hC;
    float *dA, *dB, *dC;
    int size = N * N * sizeof(float);

    hA = malloc(size);
    hB = malloc(size);
    hC = malloc(size);
    for (int i = 0; i < N * N; i++) {
        hA[i] = 2.0f;
        hB[i] = 3.0f;
    }

    hipMalloc(&dA, size);
    hipMalloc(&dB, size);
    hipMalloc(&dC, size);

    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    sgemm<<<gridSize, blockSize>>>(dA, dB, dC, N, N, N);

    hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N*N; i++) {
        if (hC[i] != 6.0f * N) {
            printf("Error: incorrect result\n");
            break;
        }
    }
    printf("Success: correct result\n");

    free(hA);
    free(hB);
    free(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}


#include <hip/hip_runtime.h>
__global__ void reduceSumKernel(float *input, float *output, int n)
{
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? input[i] : 0;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        output[blockIdx.x] = sdata[0];
    }
}


int main() {
    int n = 100000;
    float* input;
    input = (float*) malloc(n * sizeof(float));

    for (int i = 0; i < n; i++) {
        input[i] = 2.3f;
    }

    float *d_input, *d_output;

    hipMalloc((void **)&d_input, n * sizeof(float));
    hipMemcpy(d_input, input, n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipMalloc((void **)&d_output, blocksPerGrid * sizeof(float));

    reduceSumKernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(d_input, d_output, n);

    float *output = (float *)malloc(blocksPerGrid * sizeof(float));
    hipMemcpy(output, d_output, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

    float result = 0;
    for (int i = 0; i < blocksPerGrid; i++)
    {
        result += output[i];
    }

    hipFree(d_input);
    hipFree(d_output);
    free(output);
    return 0;
}

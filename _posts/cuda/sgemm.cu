#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#define BLOCK_WIDTH 16
using namespace std;


__global__ void MatrixMulKernel(int m, int n, int k, float *A,float  *B, float *C)
{
	int Row=blockIdx.y*blockDim.y+threadIdx.y;
	int Col=blockIdx.x*blockDim.x+threadIdx.x;

	if((Row<m) && (Col<k))
	{
		float cur =0.0;
		for(int i=0;i<n;++i)
		     cur += A[Row*n+i]*B[Col+i*k];
		C[Row*k+Col]= cur;
	}
}


__global__ void MatrixMulKernel(int m, int n, int k, float *A,float  *B, float *C)
{
	 //申请共享内存，存在于每个block中
	__shared__ float ds_A[BLOCK_WIDTH][BLOCK_WIDTH];
	__shared__ float ds_B[BLOCK_WIDTH][BLOCK_WIDTH];
	//简化坐标记法,出现下面6个表示的地方就是并行的地方。
	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;
	//确定结果矩阵中的行和列
	int Row = by * BLOCK_WIDTH + ty;
	int Col = bx * BLOCK_WIDTH + tx;
	float val = 0;

 	//循环读入A,B瓦片，计算结果矩阵，分阶段进行计算
	for (int t=0; t < (n-1) / BLOCK_WIDTH + 1; ++t)
	{
		//将A,B矩阵瓦片化的结果放入shared memory中，每个线程加载相应于C元素的A/B矩阵元素
		if (Row < m && t * BLOCK_WIDTH + tx < n)
		    ds_A[tx][ty] = A[Row * n + t*BLOCK_WIDTH+tx];
		else
			ds_A[tx][ty] = 0.0;

		if (t * BLOCK_WIDTH + ty < n && Col < k)
            ds_B[tx][ty] = B[(t*BLOCK_WIDTH + ty) * k + Col];
		else
			ds_B[tx][ty] = 0.0;

		__syncthreads();
		for (int i = 0; i < BLOCK_WIDTH; ++i)
            val += ds_A[i][ty] * ds_B[tx][i];

		__syncthreads();
 		if(Row < m && Col < k)
			C[Row*k+Col]=val;
	}
}


int main()
{
    float A[6] = { 11.4, 24, 33.5,   45, 55 ,32.4 }; //2×3的矩阵
	float B[12] = {12,43,22.4, 31.3,  12,324,23,12,  44.4,23.4,65.3,73};//3×4的矩阵
    float C[8] = { 0 }; //2×4的结果矩阵
	int m=2,n=3,k=4;

	int size = sizeof(float);
	float *d_a;
	float *d_b;
	float *d_c;
	hipMalloc((void**)&d_a,m*n*size);
	hipMalloc((void**)&d_b,n*k*size);
	hipMalloc((void**)&d_c,m*k*size);

	hipMemcpy(d_a, A, size*6, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size*12, hipMemcpyHostToDevice);
	hipMemcpy(d_c, C, size*8, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH,1);
	dim3 dimGrid((k-1)/BLOCK_WIDTH+1,(m-1)/BLOCK_WIDTH+1,1);

	MatrixMulKernel<<<dimGrid,dimBlock>>>(m,n,k,d_a,d_b,d_c);

	hipMemcpy(C, d_c, size*8, hipMemcpyDeviceToHost);

    for (int i=0;i<8;i++)
	{
		cout<<C[i]<<endl;
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

    return 0;
}

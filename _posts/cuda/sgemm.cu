
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1024;

__global__ void matrixMultiply(float* A, float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += A[i * n + k] * B[k * n + j];
        }
        C[i * n + j] = sum;
    }
}


int main() {
    float *hA, *hB, *hC;
    float *dA, *dB, *dC;
    int size = N * N * sizeof(float);

    // 分配内存并初始化矩阵
    hA = (float*)malloc(size);
    hB = (float*)malloc(size);
    hC = (float*)malloc(size);
    for (int i = 0; i < N * N; i++) {
        hA[i] = 1.0f;
        hB[i] = 2.0f;
    }

    // 在GPU上分配内存
    hipMalloc(&dA, size);
    hipMalloc(&dB, size);
    hipMalloc(&dC, size);

    // 将矩阵数据从主机(CPU)内存拷贝到GPU全局内存
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

    // 设置块(Block)和线程(Thread)的数量
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // 调用CUDA内核函数计算矩阵乘法
    matrixMultiply<<<gridSize, blockSize>>>(dA, dB, dC, N);

    // 将计算结果从GPU全局内存拷贝回主机(CPU)内存
    hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

    // 验证计算结果
    for (int i = 0; i < N * N; i++) {
        if (hC[i] != 2.0f * N) {
            printf("Error: incorrect result\n");
            break;
        }
    }
    printf("Success: correct result\n");

    // 释放内存
    free(hA);
    free(hB);
    free(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAddKernel(float* a, float* b, float* c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

void vectorAdd(float* a, float* b, float* c, int n)
{
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    vectorAddKernel<<<numBlocks, blockSize>>>(a, b, c, n);
}

int main()
{
    int n = 1000000;

    float* a = new float[n];
    float* b = new float[n];
    float* c = new float[n];

    for (int i = 0; i < n; i++)
    {
        a[i] = i;
        b[i] = i * 2;
    }

    vectorAdd(a, b, c, n);

    for (int i = 0; i < n; i++)
    {
        printf("%f\n", c[i]);
    }

    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}
